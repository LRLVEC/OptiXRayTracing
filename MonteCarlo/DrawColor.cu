#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include <OptiX/_Define.h>
#include "Define.h"
using namespace optix;
using namespace Define;
rtDeclareVariable(uint2, index, rtLaunchIndex, );
rtBuffer<float4, 2>result;
rtBuffer<float3>vertexBuffer;
rtBuffer<float3>vertexBufferIndexed;
rtBuffer<float3>normalBuffer;
rtBuffer<uint3>indexBuffer;
rtDeclareVariable(unsigned int, frame, , );
rtDeclareVariable(unsigned int, texid, , );
rtDeclareVariable(float3, materialColor, , );
rtDeclareVariable(Define::Trans, trans, , );
rtDeclareVariable(float, offset, , );
rtDeclareVariable(unsigned int, depthMax, , );
rtDeclareVariable(unsigned int, russian, , );
rtDeclareVariable(rtObject, group, , );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(Define::RayData, rayData, rtPayload, );
rtDeclareVariable(float, l, rtIntersectionDistance, );
rtDeclareVariable(float3, normal, attribute normal, );
rtDeclareVariable(float2, texcoord, attribute texcoord, );

//rtTextureSampler<uchar4, 3, hipReadModeNormalizedFloat> ahh;

RT_PROGRAM void rayAllocator()
{
	size_t2 size(result.size());
	uint2 screen = make_uint2(size.x, size.y);
	float2 ahh = random(index, screen, frame) + make_float2(index) - make_float2(size) / 2.0f;
	float4 d = make_float4(ahh, trans.z0, 0);
	Ray rayOrigin(trans.r0, normalize(trans.trans * d), CloseRay, offset);
	Define::RayData rayDataOrigin;
	rayDataOrigin.depth = 0;
	rayDataOrigin.weight = make_float3(1);
	rtTrace(group, rayOrigin, rayDataOrigin);
	if (frame)
		result[index] += make_float4(rayDataOrigin.color, 1.0f);
	else
		result[index] = make_float4(rayDataOrigin.color, 1.0f);
}
RT_PROGRAM void glassAnyHit()
{
	rtTerminateRay();
}
RT_PROGRAM void glassCloseHit()
{
	float3 answer = make_float3(0);
	if (rayData.depth < depthMax)
	{
		float3 r = make_float3(1);
		float3 t = make_float3(1);
		float n = 1.5f;
		float cosi1 = dot(ray.direction, normal);
		if (cosi1 > 0) n = 1 / n;
		float sini1 = sqrtf(1 - cosi1 * cosi1);
		float sini2 = sini1 / n;
		Ray rayNow;
		rayNow.origin = ray.origin + l * ray.direction;
		Define::RayData rayDataNow;
		bool seted(false);
		rayDataNow.depth = rayData.depth + 1;
		if (sini2 < 1)
		{
			float cosi2 = sqrtf(1 - sini2 * sini2);
			if (sini2 <= 0.02)
			{
				float ahh = 4 * n / ((n + 1) * (n + 1));
				t *= ahh;
				r *= 1 - ahh;
			}
			else
			{
				float a1 = n * fabsf(cosi1) + cosi2;
				float a2 = fabsf(cosi1) + n * cosi2;
				r *= (pow((n * cosi2 - fabsf(cosi1)) / a2, 2) + pow((cosi2 - n * fabsf(cosi1)) / a1, 2)) / 2;
				t *= 2 * cosi2 * (1 / pow(a1, 2) + 1 / pow(a2, 2)) * n * fabsf(cosi1);
			}
			rayDataNow.weight = rayData.weight * t;
			if (rayDataNow.weight.x + rayDataNow.weight.y + rayDataNow.weight.z > 0.01)
			{
				rayNow.direction = (ray.direction + (n * copysignf(cosi2, cosi1) - cosi1) * normal) / n;
				rayNow.tmin = offset;
				rayNow.tmax = RT_DEFAULT_MAX;
				seted = true;
				rtTrace(group, rayNow, rayDataNow);
				answer += rayDataNow.color * t;
			}
		}
		else
			r = make_float3(1);
		rayDataNow.weight = rayData.weight * r;
		if (rayDataNow.weight.x + rayDataNow.weight.y + rayDataNow.weight.z > 0.01)
		{
			rayNow.direction = ray.direction - 2 * cosi1 * normal;
			rtTrace(group, rayNow, rayDataNow);
			answer += rayDataNow.color * r;
		}
	}
	rayData.color = answer;
}
RT_PROGRAM void diffuseAnyHit()
{
	rtTerminateRay();
}
RT_PROGRAM void diffuseCloseHIt()
{
	float3 answer = make_float3(0);
	if (rayData.depth < depthMax)
	{
		float k(1);
		float2 seed(make_float2(ray.origin.y - sqrtf(frame), ray.direction.z + sqrtf(frame)));
		if (rayData.depth > russian)
		{
			if (random(seed) < 0.2f) { rayData.color = answer; return; }
			else k /= 0.8f;
		}
		float cosi1 = dot(ray.direction, normal);
		Ray rayNow;
		rayNow.origin = ray.origin + l * ray.direction;
		rayNow.tmin = offset;
		rayNow.tmax = RT_DEFAULT_MAX;
		Define::RayData rayDataNow;
		rayDataNow.depth = rayData.depth + 1;
		rayNow.direction = randomDirectionCosN(cosi1 <= 0 ? normal : -normal, 1, seed);
		rtTrace(group, rayNow, rayDataNow);
		answer += rayDataNow.color * materialColor * k;
	}
	rayData.color = answer;
}
RT_PROGRAM void miss()
{
	rayData.color = make_float3(rtTexCubemap<float4>(texid, ray.direction.x, ray.direction.y, ray.direction.z));
}
RT_PROGRAM void exception()
{
	result[index] = make_float4(1.0f, 0.0f, 0.0f, 1.0f);
}
RT_PROGRAM void attrib()
{
	unsigned int id = rtGetPrimitiveIndex();
	float3 p0 = vertexBuffer[3 * id];
	float3 p1 = vertexBuffer[3 * id + 1];
	float3 p2 = vertexBuffer[3 * id + 2];
	float3 d1 = p1 - p0;
	float3 d2 = p2 - p0;
	texcoord = rtGetTriangleBarycentrics();
	normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, normalize(cross(d1, d2))));
}
RT_PROGRAM void attribIndexed()
{
	uint3 id = indexBuffer[rtGetPrimitiveIndex()];
	texcoord = rtGetTriangleBarycentrics();
	normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, normalize(
		texcoord.x * normalBuffer[id.y] +
		texcoord.y * normalBuffer[id.z] +
		(1 - texcoord.x - texcoord.y) * normalBuffer[id.x])));
}