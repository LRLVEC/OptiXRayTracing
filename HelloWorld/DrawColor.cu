#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtBuffer<float4, 2>   result;

rtDeclareVariable(float3, color, , );

RT_PROGRAM void drawColor()
{
	result[launch_index] = make_float4(
		color.x + (float)launch_index.x / result.size().x,
		color.y + (float)launch_index.y / result.size().y,
		color.z + (float)launch_index.x / result.size().x, 0.f);
}
