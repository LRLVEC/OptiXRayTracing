#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include <OptiX/_Define.h>
#include "Define.h"
using namespace optix;

rtDeclareVariable(uint2, index, rtLaunchIndex, );
rtBuffer<float4, 2>result;
rtBuffer<float3>vertexBuffer;
rtDeclareVariable(unsigned int, frame, , );
rtDeclareVariable(float3, background, , );
rtDeclareVariable(float3, materialColor, , );
rtDeclareVariable(Define::Trans, trans, , );
rtDeclareVariable(float, offset, , );
rtDeclareVariable(rtObject, group, , );
rtDeclareVariable(Define::RayData, rayData, rtPayload, );
rtDeclareVariable(float3, normal, attribute normal, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );

RT_PROGRAM void rayAllocator()
{
	size_t2 size(result.size());
	uint2 screen = make_uint2(size.x, size.y);
	float2 ahh = random(index, screen, frame) + make_float2(index) - make_float2(size) / 2.0f;
	float4 d = make_float4(ahh, trans.z0, 0);
	Ray ray(trans.r0, normalize(trans.trans * d), CloseRay, offset);
	Define::RayData rayData;
	rayData.depth = 0;
	rtTrace(group, ray, rayData);
	if (frame)
		result[index] += make_float4(rayData.color, 1.0f);
	else
		result[index] = make_float4(rayData.color, 1.0f);
}
RT_PROGRAM void anyHit()
{
	rtTerminateRay();
}
RT_PROGRAM void closeHit()
{
	rayData.color = materialColor * normalize(normal + make_float3(1, 1, 1));
}
RT_PROGRAM void miss()
{
	rayData.color = background;
}
RT_PROGRAM void exception()
{
	result[index] = make_float4(1.0f, 0.0f, 0.0f, 0.0f);
}
RT_PROGRAM void attrib()
{
	unsigned int id = rtGetPrimitiveIndex();
	float3 p0 = vertexBuffer[3 * id];
	float3 p1 = vertexBuffer[3 * id + 1];
	float3 p2 = vertexBuffer[3 * id + 2];
	float3 d1 = p1 - p0;
	float3 d2 = p2 - p0;
	normal = normalize(cross(d1, d2));
	texcoord = make_float3(rtGetTriangleBarycentrics());
}