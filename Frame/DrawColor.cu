#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include <OptiX/_Define.h>
using namespace optix;

rtDeclareVariable(uint2, index, rtLaunchIndex, );
rtBuffer<float4, 2>result;
rtBuffer<float3> vertices;
rtDeclareVariable(unsigned int, frame, , );
rtDeclareVariable(float3, background, , );
rtDeclareVariable(float3, materialColor, , );
rtDeclareVariable(Define::Trans, trans, , );
rtDeclareVariable(float, offset, , );
rtDeclareVariable(rtObject, group, , );
rtDeclareVariable(Define::RayData, rayData, rtPayload, );

RT_PROGRAM void rayAllocator()
{
	size_t2 size(result.size());
	uint2 screen = make_uint2(size.x, size.y);
	float2 ahh = random(index, screen, frame) + make_float2(index) - make_float2(size) / 2.0f;
	float4 d = make_float4(ahh, trans.z0, 0);
	Ray ray(trans.r0, normalize(trans.trans * d), CloseRay, offset);
	Define::RayData rayData;
	rayData.depth = 0;
	rtTrace(group, ray, rayData);
	if (frame)
		result[index] += make_float4(rayData.color, 1.0f);
	else
		result[index] = make_float4(rayData.color, 1.0f);
}
RT_PROGRAM void anyHit()
{
	rtTerminateRay();
}
RT_PROGRAM void closeHit()
{
	rayData.color = materialColor;
}
RT_PROGRAM void miss()
{
	rayData.color = background;
}
RT_PROGRAM void exception()
{
	result[index] = make_float4(1.0f, 0.0f, 0.0f, 0.0f);
}