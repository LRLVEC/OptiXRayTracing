#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "Define.h"
using namespace optix;

rtDeclareVariable(uint2, index, rtLaunchIndex, );
rtBuffer<float4, 2>result;
rtBuffer<float3> vertices;
rtDeclareVariable(unsigned int, frame, , );
rtDeclareVariable(Define::Color, background, , );
rtDeclareVariable(float3, materialColor, , );
rtDeclareVariable(Define::Eye, eye, , );
rtDeclareVariable(float, offset, , );
rtDeclareVariable(rtObject, group, , );
rtDeclareVariable(Define::RayData, rayData, rtPayload, );

RT_PROGRAM void rayAllocator()
{
	size_t2 screen = result.size();
	float3 d = make_float3(
		float(index.x * 2) - float(screen.x),
		float(index.y * 2) - float(screen.y),
		eye.z0);
	optix::Ray ray(eye.r0, normalize(d), CloseRay, offset);
	Define::RayData rayData;
	rayData.depth = 0;
	rtTrace(group, ray, rayData);
	result[index] = make_float4(rayData.color, 1.0f);
}
RT_PROGRAM void anyHit()
{
	rtTerminateRay();
}
RT_PROGRAM void closeHit()
{
	rayData.color = make_float3(0.0f, 1.0f, 0.0f);
}
RT_PROGRAM void miss()
{
	rayData.color = background;
}
RT_PROGRAM void exception()
{
	result[index] = make_float4(1.0f, 0.0f, 0.0f, 0.0f);
}