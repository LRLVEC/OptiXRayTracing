#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include <OptiX/_Define.h>
#include "Define.h"
using namespace optix;
using namespace Define;
rtDeclareVariable(uint2, index, rtLaunchIndex, );
rtBuffer<float4, 2>result;
rtBuffer<float3>vertexBuffer;
rtBuffer<float3>vertexBufferIndexed;
rtBuffer<float3>normalBuffer;
rtBuffer<uint3>indexBuffer;
rtDeclareVariable(unsigned int, frame, , );
rtDeclareVariable(unsigned int, texid, , );
rtDeclareVariable(float3, materialColor, , );
rtDeclareVariable(Define::Trans, trans, , );
rtDeclareVariable(float, offset, , );
rtDeclareVariable(unsigned int, depthMax, , );
rtDeclareVariable(unsigned int, russian, , );
rtDeclareVariable(rtObject, group, , );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(Define::RayData, rayData, rtPayload, );
rtDeclareVariable(float, l, rtIntersectionDistance, );
rtDeclareVariable(float3, normal, attribute normal, );
rtDeclareVariable(float2, texcoord, attribute texcoord, );

//rtTextureSampler<uchar4, 3, hipReadModeNormalizedFloat> ahh;

RT_PROGRAM void rayAllocator()
{
	size_t2 size(result.size());
	uint2 screen = make_uint2(size.x, size.y);
	float2 ahh = random(index, screen, frame) + make_float2(index) - make_float2(size) / 2.0f;
	float4 d = make_float4(ahh, trans.z0, 0);
	Ray rayOrigin(trans.r0, normalize(trans.trans * d), CloseRay, offset);
	Define::RayData rayDataOrigin;
	rayDataOrigin.depth = 0;
	//rayDataOrigin.weight = make_float3(1);
	rtTrace(group, rayOrigin, rayDataOrigin);
	if (frame)
		result[index] += make_float4(rayDataOrigin.color, 1.0f);
	else
		result[index] = make_float4(rayDataOrigin.color, 1.0f);
}
RT_PROGRAM void metalAnyHit()
{
	rtTerminateRay();
}
RT_PROGRAM void metalCloseHit()
{
	float3 answer = make_float3(0);
	if (rayData.depth < depthMax)
	{
		Ray rayNow;
		rayNow.origin = ray.origin + l * ray.direction;
		Define::RayData rayDataNow;
		rayDataNow.depth = rayData.depth + 1;
		rayNow.direction = ray.direction - 2 * dot(ray.direction, normal) * normal;
		rayNow.tmin = offset;
		rayNow.tmax = RT_DEFAULT_MAX;
		rtTrace(group, rayNow, rayDataNow);
		answer = rayDataNow.color * materialColor;
	}
	rayData.color = answer;
}
RT_PROGRAM void glassAnyHit()
{
	rtTerminateRay();
}
RT_PROGRAM void glassCloseHit()
{
	float3 answer = make_float3(0);
	float decay = 0.1;
	if (rayData.depth < 15)
	{
		float4 r = make_float4(1);
		float4 t = make_float4(1);
		float n = 1.5f;
		float cosi1 = dot(ray.direction, normal);
		if (cosi1 > 0) n = 1 / n;
		float sini1 = sqrtf(1 - cosi1 * cosi1);
		float sini2 = sini1 / n;
		Ray rayNow;
		rayNow.origin = ray.origin + l * ray.direction;
		Define::RayData rayDataNow;
		bool seted(false);
		rayDataNow.depth = rayData.depth + 1;
		if (sini2 < 1)
		{
			float cosi2 = sqrtf(1 - sini2 * sini2);
			if (sini2 <= 0.02)
			{
				float ahh = 4 * n / ((n + 1) * (n + 1));
				t.w *= ahh;
				r.w *= 1 - ahh;
			}
			else
			{
				float a1 = n * fabsf(cosi1) + cosi2;
				float a2 = fabsf(cosi1) + n * cosi2;
				r.w *= (pow((n * cosi2 - fabsf(cosi1)) / a2, 2) + pow((cosi2 - n * fabsf(cosi1)) / a1, 2)) / 2;
				t.w *= 2 * cosi2 * (1 / pow(a1, 2) + 1 / pow(a2, 2)) * n * fabsf(cosi1);
			}
			rayNow.direction = (ray.direction + (n * copysignf(cosi2, cosi1) - cosi1) * normal) / n;
			rayNow.tmin = offset;
			rayNow.tmax = RT_DEFAULT_MAX;
			seted = true;
			rtTrace(group, rayNow, rayDataNow);
			if (cosi1 > 0) { float s(expf(-decay * l)); t.w *= s; r.w *= s; }
			answer += rayDataNow.color * make_float3(t) * t.w;
		}
		else
			r.w = expf(-decay * l);
		rayNow.direction = ray.direction - 2 * cosi1 * normal;
		rtTrace(group, rayNow, rayDataNow);
		answer += rayDataNow.color * make_float3(r) * r.w;
	}
	rayData.color = answer;
}
RT_PROGRAM void diffuseAnyHit()
{
	rtTerminateRay();
}
RT_PROGRAM void diffuseCloseHIt()
{
	float3 answer = make_float3(0);
	if (rayData.depth < depthMax)
	{
		float k(1);
		float2 seed(make_float2(ray.origin.y - sqrtf(frame), ray.direction.z + sqrtf(frame)));
		if (rayData.depth > russian)
		{
			if (random(seed) < 0.2f) { rayData.color = answer; return; }
			else k /= 0.8f;
		}
		float cosi1 = dot(ray.direction, normal);
		Ray rayNow;
		rayNow.origin = ray.origin + l * ray.direction;
		rayNow.tmin = offset;
		rayNow.tmax = RT_DEFAULT_MAX;
		Define::RayData rayDataNow;
		rayDataNow.depth = rayData.depth + 1;
		rayNow.direction = randomDirectionCosN(cosi1 <= 0 ? normal : -normal, 1, seed);
		rtTrace(group, rayNow, rayDataNow);
		answer += rayDataNow.color * materialColor * k;
	}
	rayData.color = answer;
}
RT_PROGRAM void scatterAnyHit()
{
	rtTerminateRay();
}
RT_PROGRAM void scatterCloseHit()
{
	float3 answer = make_float3(0);
	float3 ratio;
	float s = 0.7;
	float decay = 0.6;
	float distance;
	if (rayData.depth < depthMax)
	{
		float k(1);
		float3 seed(make_float3(
			ray.origin.y - sqrtf(frame),
			ray.direction.z + sqrtf(frame),
			ray.direction.x + sqrtf(frame)
		));
		float gg(random(make_float2(seed)));
		if (rayData.depth > russian)
		{
			if (gg < 0.2f) { rayData.color = answer; return; }
			else k /= 0.8f;
		}
		float cosi1 = dot(ray.direction, normal);
		Ray rayNow;
		rayNow.tmin = offset;
		rayNow.tmax = RT_DEFAULT_MAX;
		Define::RayData rayDataNow;
		rayDataNow.depth = rayData.depth + 1;
		gg = random(make_float2(seed.y, seed.z));
		if (cosi1 > 0 && gg > expf(-l * s))
		{
			float4 ahh(randomScatter(ray.direction, l, s, seed));
			distance = ahh.w;
			rayNow.origin = ray.origin + ahh.w * ray.direction;
			rayNow.direction = make_float3(ahh);
			ratio = Define::scatterRatio;
			k *= expf(-decay * distance);
		}
		else
		{
			distance = l;
			rayNow.origin = ray.origin + l * ray.direction;
			rayNow.direction = ray.direction;
			ratio = { 1,1,1 };
			if (cosi1 > 0)k *= expf(-decay * distance);
		}
		rtTrace(group, rayNow, rayDataNow);
		answer += rayDataNow.color * materialColor * ratio * k;
	}
	rayData.color = answer;
}
RT_PROGRAM void miss()
{
	rayData.color = make_float3(rtTexCubemap<float4>(texid, ray.direction.x, ray.direction.y, ray.direction.z));
	//if (ray.direction.x > 0.9)
	//	rayData.color = { 50,50,50 };//
	//else
	//	rayData.color = { 0 };
}
RT_PROGRAM void exception()
{
	result[index] = make_float4(1.0f, 0.0f, 0.0f, 1.0f);
}
RT_PROGRAM void attrib()
{
	unsigned int id = rtGetPrimitiveIndex();
	float3 p0 = vertexBuffer[3 * id];
	float3 p1 = vertexBuffer[3 * id + 1];
	float3 p2 = vertexBuffer[3 * id + 2];
	float3 d1 = p1 - p0;
	float3 d2 = p2 - p0;
	texcoord = rtGetTriangleBarycentrics();
	normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, normalize(cross(d1, d2))));
}
RT_PROGRAM void attribIndexed()
{
	uint3 id = indexBuffer[rtGetPrimitiveIndex()];
	texcoord = rtGetTriangleBarycentrics();
	normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, normalize(
		texcoord.x * normalBuffer[id.y] +
		texcoord.y * normalBuffer[id.z] +
		(1 - texcoord.x - texcoord.y) * normalBuffer[id.x])));
}