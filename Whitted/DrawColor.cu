#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include <OptiX/_Define.h>
#include "Define.h"
using namespace optix;

rtDeclareVariable(uint2, index, rtLaunchIndex, );
rtBuffer<float4, 2>result;
rtBuffer<float3>vertexBuffer;
rtDeclareVariable(unsigned int, frame, , );
rtDeclareVariable(unsigned int, texid, , );
//rtDeclareVariable(float3, background, , );
rtDeclareVariable(float3, materialColor, , );
rtDeclareVariable(Define::Trans, trans, , );
rtDeclareVariable(float, offset, , );
rtDeclareVariable(rtObject, group, , );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t, rtIntersectionDistance, );
rtDeclareVariable(Define::RayData, rayData, rtPayload, );
rtDeclareVariable(float3, normal, attribute normal, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );

//rtTextureSampler<uchar4, 3, hipReadModeNormalizedFloat> ahh;

RT_PROGRAM void rayAllocator()
{
	size_t2 size(result.size());
	uint2 screen = make_uint2(size.x, size.y);
	float2 ahh = random(index, screen, frame) + make_float2(index) - make_float2(size) / 2.0f;
	float4 d = make_float4(ahh, trans.z0, 0);
	Ray rayOrigin(trans.r0, normalize(trans.trans * d), CloseRay, offset);
	Define::RayData rayDataOrigin;
	rayDataOrigin.depth = 0;
	rayDataOrigin.color = make_float3(0);
	rayDataOrigin.weight = make_float3(1);
	rtTrace(group, rayOrigin, rayDataOrigin);
	if (frame)
		result[index] += make_float4(rayDataOrigin.color, 1.0f);
	else
		result[index] = make_float4(rayDataOrigin.color, 1.0f);
}
RT_PROGRAM void anyHit()
{
	rtTerminateRay();
}
RT_PROGRAM void closeHit()
{
	if (rayData.depth < 3)
	{
		Ray rayRefl(
			ray.origin + t * ray.direction,
			ray.direction - 2 * dot(normal, ray.direction) * normal,
			CloseRay, offset);
		Define::RayData rayDataRefl = rayData;
		++rayDataRefl.depth;
		rtTrace(group, rayRefl, rayDataRefl);
		rayData.color += rayDataRefl.color * 0.9f;
	}
}
RT_PROGRAM void miss()
{
	rayData.color = make_float3(rtTexCubemap<float4>(texid, ray.direction.x, ray.direction.y, ray.direction.z));
}
RT_PROGRAM void exception()
{
	result[index] = make_float4(1.0f, 0.0f, 0.0f, 0.0f);
}
RT_PROGRAM void attrib()
{
	unsigned int id = rtGetPrimitiveIndex();
	float3 p0 = vertexBuffer[3 * id];
	float3 p1 = vertexBuffer[3 * id + 1];
	float3 p2 = vertexBuffer[3 * id + 2];
	float3 d1 = p1 - p0;
	float3 d2 = p2 - p0;
	normal = normalize(cross(d1, d2));
	texcoord = make_float3(rtGetTriangleBarycentrics());
}