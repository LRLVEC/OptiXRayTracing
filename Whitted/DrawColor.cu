#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include <OptiX/_Define.h>
#include "Define.h"
using namespace optix;

rtDeclareVariable(uint2, index, rtLaunchIndex, );
rtBuffer<float4, 2>result;
rtBuffer<float3>vertexBuffer;
rtBuffer<float3>normalBuffer;
rtBuffer<uint3>indexBuffer;
rtDeclareVariable(unsigned int, frame, , );
rtDeclareVariable(unsigned int, texid, , );
//rtDeclareVariable(float3, background, , );
rtDeclareVariable(float3, materialColor, , );
rtDeclareVariable(Define::Trans, trans, , );
rtDeclareVariable(float, offset, , );
rtDeclareVariable(unsigned int, depthMax, , );
rtDeclareVariable(rtObject, group, , );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(Define::RayData, rayData, rtPayload, );
rtDeclareVariable(float, l, rtIntersectionDistance, );
rtDeclareVariable(float3, normal, attribute normal, );
rtDeclareVariable(float2, texcoord, attribute texcoord, );

//rtTextureSampler<uchar4, 3, hipReadModeNormalizedFloat> ahh;

RT_PROGRAM void rayAllocator()
{
	size_t2 size(result.size());
	uint2 screen = make_uint2(size.x, size.y);
	float2 ahh = random(index, screen, frame) + make_float2(index) - make_float2(size) / 2.0f;
	float4 d = make_float4(ahh, trans.z0, 0);
	Ray rayOrigin(trans.r0, normalize(trans.trans * d), CloseRay, offset);
	Define::RayData rayDataOrigin;
	rayDataOrigin.depth = 0;
	rayDataOrigin.weight = make_float3(1);
	rtTrace(group, rayOrigin, rayDataOrigin);
	if (frame)
		result[index] += make_float4(rayDataOrigin.color, 1.0f);
	else
		result[index] = make_float4(rayDataOrigin.color, 1.0f);
}
RT_PROGRAM void anyHit()
{
	rtTerminateRay();
}
RT_PROGRAM void closeHit()
{
	float3 answer = make_float3(0);
	if (rayData.depth < depthMax)
	{
		float3 r = make_float3(1);
		float3 t = make_float3(1);
		float n = 1.5f;
		float cosi1 = dot(ray.direction, normal);
		if (cosi1 > 0) n = 1 / n;
		float sini1 = sqrtf(1 - cosi1 * cosi1);
		float sini2 = sini1 / n;
		Ray rayNow;
		rayNow.origin = ray.origin + l * ray.direction;
		Define::RayData rayDataNow;
		rayDataNow.color = make_float3(0);
		rayDataNow.depth = rayData.depth + 1;
		if (sini2 < 1)
		{
			float cosi2 = sqrtf(1 - sini2 * sini2);
			if (sini2 <= 0.02)
			{
				float ahh = 4 * n / ((n + 1) * (n + 1));
				t *= ahh;
				r *= 1 - ahh;
			}
			else
			{
				float a1 = n * fabsf(cosi1) + cosi2;
				float a2 = fabsf(cosi1) + n * cosi2;
				r *= (pow((n * cosi2 - fabsf(cosi1)) / a2, 2) + pow((cosi2 - n * fabsf(cosi1)) / a1, 2)) / 2;
				t *= 2 * cosi2 * (1 / pow(a1, 2) + 1 / pow(a2, 2)) * n * fabsf(cosi1);
			}
			rayDataNow.weight = rayData.weight * t;
			if (rayDataNow.weight.x + rayDataNow.weight.y + rayDataNow.weight.z > 0.01)
			{
				rayNow.direction = (ray.direction + (n * copysignf(cosi2, cosi1) - cosi1) * normal) / n;
				rayNow.tmin = offset;
				rayNow.tmax = RT_DEFAULT_MAX;
				rtTrace(group, rayNow, rayDataNow);
				rayNow.origin = ray.origin + l * ray.direction;
				answer += rayDataNow.color * t;
			}
		}
		else
			r = make_float3(1);
		rayDataNow.weight = rayData.weight * t;
		if (rayDataNow.weight.x + rayDataNow.weight.y + rayDataNow.weight.z > 0.01)
		{
			rayNow.direction = ray.direction - 2 * cosi1 * normal;
			rayNow.tmin = offset;
			rayNow.tmax = RT_DEFAULT_MAX;
			rayDataNow.color = make_float3(0);
			rtTrace(group, rayNow, rayDataNow);
			answer += rayDataNow.color * r;
		}
	}
	rayData.color = answer;
}
RT_PROGRAM void miss()
{
	rayData.color = make_float3(rtTexCubemap<float4>(texid, ray.direction.x, ray.direction.y, ray.direction.z));
}
RT_PROGRAM void exception()
{
	result[index] = make_float4(1.0f, 0.0f, 0.0f, 1.0f);
}
RT_PROGRAM void attrib()
{
	uint3 id = indexBuffer[rtGetPrimitiveIndex()];
	//float3 p0 = vertexBuffer[3 * id];
	//float3 p1 = vertexBuffer[3 * id + 1];
	//float3 p2 = vertexBuffer[3 * id + 2];
	//float3 d1 = p1 - p0;
	//float3 d2 = p2 - p0;
	texcoord = rtGetTriangleBarycentrics();
	normal = normalize(
		texcoord.x * normalBuffer[id.y] +
		texcoord.y * normalBuffer[id.z] +
		(1 - texcoord.x - texcoord.y) * normalBuffer[id.x]);
}